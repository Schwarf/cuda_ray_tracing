#include "hip/hip_runtime.h"
//
// Created by andreas on 01.11.21.
//

#include "cuda_implementation/miscellaneous/templates/n_tuple.cuh"
#include "cuda_implementation/rays/ray.cuh"
#include "cuda_implementation/rays/hit_record.cuh"
#include "cuda_implementation/objects/sphere.cuh"
#include "cuda_implementation/objects/object_list.cuh"
#include <fstream>
#include "cuda_implementation/materials/material.cuh"
#include "cuda_implementation/rays/ray_interactions.cuh"
#include <iostream>
#include "cuda_implementation/miscellaneous/CUDAMemory.cuh"

#define checkCudaErrors(value) check_cuda( (value), #value, __FILE__, __LINE__)

__device__ __host__ void build_material(IMaterial * const p_material)
{
	p_material->set_specular_reflection(0.3f);
	p_material->set_diffuse_reflection(0.6);
	p_material->set_ambient_reflection(0.3);
	p_material->set_shininess(50.0);
	p_material->set_transparency(0.0001);
	p_material->set_refraction_index(1.0);
	Vector3D color = Vector3D{0.9, 0.2, 0.3};
	p_material->set_rgb_color(color);
}


__device__ __host__ void build_material2(IMaterial * const p_material)
{
	p_material->set_specular_reflection(0.5f);
	p_material->set_diffuse_reflection(0.2);
	p_material->set_ambient_reflection(0.1);
	p_material->set_shininess(50.0);
	p_material->set_transparency(0.0001);
	p_material->set_refraction_index(1.0);
	Vector3D color = Vector3D{0.3, 0.9, 0.3};
	p_material->set_rgb_color(color);
}


__global__ void create_objects(ITargetObject ** target_objects, ObjectList *object_list)
{
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		auto sphere_center = Vector3D{-3.5f, 3.5f, -15.f};
		auto sphere_radius = 1.5f;
		auto sphere_center2 = Vector3D{0.5f, -1.5f, -10.f};
		auto sphere_radius2 = 2.5f;

		Material material;
		IMaterial *p_material = &material;

		Material material2;
		IMaterial *p_material2 = &material2;

		build_material(p_material);
		build_material2(p_material2);
		auto sphere = Sphere(sphere_center, sphere_radius, p_material);
		auto sphere2 = Sphere(sphere_center2, sphere_radius2, p_material2);

		auto p_sphere = &sphere;
		auto p_sphere2 = &sphere2;
		target_objects[0] = p_sphere;
		target_objects[1] = p_sphere2;
		object_list = new ObjectList(target_objects, 2);
	}
}

__device__ __host__ Color get_pixel_color(const IRay &ray,
										  ObjectList &object_list,
										  IHitRecord &hit_record,
										  IRayInteractions & ray_interaction,
										  size_t recursion_depth)
{
	auto is_hit = object_list.hit_by_ray(ray, hit_record);
	if (!is_hit) {
		return Color{0.2, 0.7, 0.8};
	}
	recursion_depth--;
	auto diffuse_ray = Ray();
	auto specular_ray = Ray();
	ray_interaction.diffuse_scatter(hit_record, diffuse_ray);
	ray_interaction.specular_scatter(ray, hit_record, specular_ray);
//	auto reflected_color = get_pixel_color(diffuse_ray, sphere, hit_record, ray_interaction, recursion_depth);
//	auto refracted_color = get_pixel_color(specular_ray, sphere, hit_record, ray_interaction, recursion_depth);
	Color diffuse_color = hit_record.get_material()->diffuse_reflection() * hit_record.get_material()->rgb_color();
	Color white = Color{1, 1, 1};
	Color specular_color = white * hit_record.get_material()->specular_reflection();
	return diffuse_color + specular_color;
}

__global__ void render_it(Vector3D *buffer, size_t max_width, size_t max_height, ITargetObject ** object_list)
{
	//size_t width = threadIdx.x + blockIdx.x * blockDim.x;
	//size_t height = threadIdx.y + blockIdx.y * blockDim.y;

	size_t width = threadIdx.x;
	size_t height = blockIdx.x;
	if ((width >= max_width) || (height >= max_height)) {
		return;
	}
	float x_direction = float(width) - float(max_width) / 2.f;
	float y_direction = float(height) - float(max_height) / 2.f;
	float z_direction = -float(max_height + max_width) / 2.f;
	auto sphere_center = Vector3D{-3.5f, 3.5f, -15.f};
	auto sphere_radius = 1.5f;
	auto sphere_center2 = Vector3D{0.5f, -1.5f, -10.f};
	auto sphere_radius2 = 2.5f;

	Material material;
	IMaterial * p_material = & material;

	Material material2;
	IMaterial * p_material2 = & material2;

	build_material(p_material);
	build_material2(p_material2);
	auto sphere = Sphere(sphere_center, sphere_radius, p_material);
	auto sphere2 = Sphere(sphere_center2, sphere_radius2, p_material2);


	auto p_sphere = & sphere;
	auto p_sphere2 = & sphere2;
	object_list[0] = p_sphere;
	object_list[1] = p_sphere2;
	auto list = ObjectList(object_list, 2);

	Vector3D direction = Vector3D{x_direction, y_direction, z_direction}.normalize();
	Vector3D origin = Vector3D{0, 0, 0};
	auto ray = Ray(origin, direction);
	auto hit_record = HitRecord();
	auto ray_interactions = RayInteractions();
	Color pixel_color = get_pixel_color(ray, list, hit_record, ray_interactions, 2);
	size_t pixel_index = height * max_width + width;
	buffer[pixel_index] = pixel_color;

}

int main()
{
	size_t width = 1024;
	size_t height = 768;
	// Why is 32 the maximum number of threads per block
	constexpr size_t threads_per_block = 32;
	//dim3 number_of_threads(threads_per_block, threads_per_block);

	//dim3 number_of_blocks(width / threads_per_block, height / threads_per_block);
	int number_of_blocks = 768;
	int number_of_threads{1024};
	size_t buffer_size = width * height * sizeof(float3);
	std::cout << buffer_size << std::endl;
	Color *buffer;
	CUDAMemory<Color>::allocate_managed_instance(buffer, buffer_size);
	ITargetObject ** target_objects;
	//ObjectList *object_list;
	//CUDAMemory<ObjectList>::allocate_instance(object_list, 1);
	CUDAMemory<ITargetObject>::allocate_pointer_to_instance(target_objects, 2);
	//hipMallocManaged((void **)&buffer, buffer_size);
	//create_objects<<<1,1>>>(target_objects, object_list);

	render_it<<<number_of_blocks, number_of_threads>>>(buffer, width, height, target_objects);
	hipGetLastError();
	hipDeviceSynchronize();
	std::ofstream ofs;
	ofs.open("./cuda_image.ppm");
	ofs << "P6\n" << width << " " << height << "\n255\n";
	for (size_t pixel_index = 0; pixel_index < width * height; ++pixel_index) {
		for (size_t color_index = 0; color_index < 3; color_index++) {
			ofs << static_cast<char>(255 * std::max(0.f, std::min(1.f, buffer[pixel_index][color_index])));
		}
	}CUDAMemory<Color>::release(buffer);

	return 0;
}