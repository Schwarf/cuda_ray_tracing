#include "hip/hip_runtime.h"
//
// Created by andreas on 22.12.21.
//

#include "light_source_effects.cuh"

__device__ __host__ LightSourceEffects::LightSourceEffects(ILightSource **&list, size_t size):
light_source_list_(list),
number_of_light_sources_(size)
{
}


__device__ __host__ void LightSourceEffects::compute_light_source_effects(const IRay &ray,
													  IHitRecord &hit_record,
													  float &diffuse_intensity,
													  float &specular_intensity) const
{

	auto hit_normal = hit_record.hit_normal();
	auto hit_point = hit_record.hit_point();
	auto material = hit_record.get_material();
	auto light_source_ray = Ray();
	auto shadow_hit_record = HitRecord();
	for (size_t ls_index = 0; ls_index < number_of_light_sources_; ++ls_index) {
		ILightSource *light_source = light_source_list_[ls_index];
		Vector3D light_direction = (light_source->position() - hit_point).normalize();
		light_source_ray.set_direction(light_direction);
		light_source_ray.set_origin(hit_point);
		auto reflected_ray = Ray();
		specular_scatter(light_source_ray, hit_record, reflected_ray);
		diffuse_intensity += light_source->intensity() * fmaxf(0.f, light_direction * hit_normal);
		specular_intensity +=
			powf(fmaxf(0.f, reflected_ray.direction_normalized() * ray.direction_normalized()), material->shininess())
				* light_source->intensity();
	}

}